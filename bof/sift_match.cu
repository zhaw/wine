#include "hip/hip_runtime.h"
#include "tmwtypes.h"

__global__ void sift_match (int *index, const uint8_T *vn,
        const uint8_T *vm, const int n, const int m, const double thres)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int diff;
    int tmp;
    int local_vm[128];
    for (int j = 0; j < 128; j++) local_vm[j] = vm[idx*128+j]; 
    if (idx < m)
    {
        int second_best = 9999999;
        int best_v = 999999999;
        int best_i = -1;
        for (int i = 0; i < n; i++)
        {
            tmp = 0;
            for (int j = 0; j < 128; j++)
            {
                diff = vn[i*128+j] - local_vm[j];
                tmp += diff * diff;
                if ( tmp > second_best ) break;
            }
            if ( tmp < best_v )
            {
                second_best = best_v;
                best_v = tmp;
                best_i = i;
            }
            else if ( tmp < second_best ) second_best = tmp;
        }
        if ( best_v * thres < second_best )
            index[idx] = best_i;
        else
            index[idx] = -1;
/*        int best_v = 99999999;
        int best_i = -1;
        for (int i = 0; i < n; i++)
        {
            if (dist[i] < best_v)
            {
                best_i = -1;
                if (dist[i] * thres < best_v) best_i = i;
                best_v = dist[i];
            }
        }
        index[idx] = best_i;
*/
    }
}
