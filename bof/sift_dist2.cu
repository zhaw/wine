#include "hip/hip_runtime.h"
#include <math.h>
#include "tmwtypes.h"

__global__ void sift_dist2 ( int * d, const uint8_T * vn, const uint8_T * vm, const int n, const int m)
{
    // d:  n * m
    // v1: 128 x n
    // v2: 128 x m
    int diff;
    int idx1 = blockIdx.x;
    int idx2 = blockIdx.y;
    int i = threadIdx.x;
    diff = vm[idx2*128+i] - vn[idx1*128+i];
    atomicAdd(&d[idx2*n+idx1], diff*diff);
}
