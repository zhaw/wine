#include "hip/hip_runtime.h"
#include <math.h>
#include "tmwtypes.h"

__global__ void sift_dist ( int * d, const uint8_T * vn, const uint8_T * vm, const int n, const int m)
{
    // d:  n * m
    // v1: 128 x n
    // v2: 128 x m
    int diff;
    int idx1 = blockIdx.x*blockDim.x + threadIdx.x;
    int idx2 = blockIdx.y*blockDim.y + threadIdx.y;
    if ( idx1 < n && idx2 < m )
    {
        for (int i = 0; i < 128; i++)
        {
            diff = vn[idx1*128+i] - vm[idx2*128+i];
            d[idx2*n+idx1] += diff * diff;
        }
    }
}
