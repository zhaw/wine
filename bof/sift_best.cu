#include "hip/hip_runtime.h"
#include <math.h>
#include "tmwtypes.h"

__global__ void sift_best (int * index, const int * dist, const int n, const int m, const double thres)
{
    // index: m1+m2+...
    // dist:  n*(m1+m2+...)
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < m)
    {
        int best_v = 99999999;
        int best_i = -1;
        for (int i = 0; i < n; i++)
        {
            if (dist[idx*n+i] < best_v)
            {
                best_i = -1;
                if (dist[idx*n+i] * thres < best_v) best_i = i;
                best_v = dist[idx*n+i];
            }
        }
        index[idx] = best_i;
    }
}
