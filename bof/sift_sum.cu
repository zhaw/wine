#include "hip/hip_runtime.h"
#include <math.h>
#include "tmwtypes.h"

__global__ void sift_sum ( int *s, const int *d , const int n, const int m)
{
    // d: 128 x n x m
    // s: n x m
    if (blockIdx.x*blockDim.x + threadIdx.x < n
        && blockIdx.y*blockDim.y + threadIdx.y < m)
        {
            int i = 0;
            int a = blockIdx.x * blockDim.x + threadIdx.x;
            int b = blockIdx.y * blockDim.y + threadIdx.y;
            int idx = a + b*n;
            for (i = 128*idx; i < 128+128*idx; i++)
                s[idx] += d[i];
        }
}
